#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <SDL/SDL.h>

#define pot_const 1e-6
#define BLOCKSIZE 16
#define OFFSET 10
#define AUXSIGMA 3

typedef struct coord_t {float x, y, m;} coord_t;

__device__
coord_t potential(coord_t contrib, coord_t a, coord_t b)
{
  contrib.x += pot_const * b.m / powf( sqrtf( powf((a.x-b.x), 2) + powf(a.y-b.y, 2) ), 3)*(b.x-a.x);
  contrib.y += pot_const * b.m / powf( sqrtf( powf((a.x-b.x), 2) + powf(a.y-b.y, 2) ), 3)*(b.y-a.y);
  return contrib;
}


__global__ 
void init_coordinates_n_masses(coord_t * bodies_1, coord_t * bodies_2, hiprandState_t * states, size_t X, 
                               float max_X, float max_Y, float min_m, float max_m, int seed)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  hiprand_init(seed, j*X+i, OFFSET, &states[j*X+i]);
  bodies_1[j*X+i].x = hiprand_uniform(&states[j*X+i])*max_X;
  bodies_1[j*X+i].y = hiprand_uniform(&states[j*X+i])*max_Y;
  bodies_1[j*X+i].m = min_m+ hiprand_uniform(&states[j*X+i]) * (max_m-min_m);
  //bodies_1[j*X+i].m = min_m+ hiprand_log_normal(&states[j*X+i], (max_m+min_m)/2, (max_m-min_m)/AUXSIGMA);
  bodies_2[j*X+i].m = bodies_1[j*X+i].m;
}

__global__ 
void update_bodies(coord_t * bodies_in, coord_t * bodies_out, size_t X, size_t Y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  
  coord_t contrib = {0,0};
  for(int h=0; h<Y; h++)
    for(int w=0; w<X; w++)
      if ( h != j && w != i)
        contrib = potential(contrib, bodies_in[j*X+i], bodies_in[h*X+w]);
  bodies_out[j*X+i].x = bodies_in[j*X+i].x + contrib.x;
  bodies_out[j*X+i].y = bodies_in[j*X+i].y + contrib.y;

}

void update_pixels(coord_t * h_bodies, Uint32 * pixels, float max_X, float max_Y, float min_m, float max_m, 
                   size_t X, size_t Y, size_t nb_pts_X, size_t nb_pts_Y)
{
  size_t px, py;
  for (int y=0; y<Y; y++)
    for (int x=0; x<X; x++)
    {
      px = ceil( nb_pts_X * h_bodies[y*X+x].x / max_X);
      py = ceil( nb_pts_Y * h_bodies[y*X+x].y / max_Y);
      if (0<py && py<nb_pts_Y && 0<px && px< nb_pts_X)
        pixels[py*nb_pts_X+px] = floor( (16711680-65280) * (h_bodies[y*X+x].m - min_m) / max_m + 65280);
    }
}

int main(int argc, char * argv[])
{
  if (argc != 8)
  {
    printf("nb bodies, nb_pts_X, nb_pts_Y, max_X, max_Y, min_m, max_m\n");
    return EXIT_SUCCESS;
  }

  size_t N = atoi(argv[1]),
         nb_pts_X  =  atoi(argv[2]),
         nb_pts_Y = atoi(argv[3]);
  float max_X = atof(argv[4]),
        max_Y = atof(argv[5]),
        min_m = atof(argv[6]),
        max_m = atof(argv[7]);

  size_t X, Y; 
  X = ceil(sqrt(N));
  Y = ceil((float) N/X);

  printf("X, Y = %d, %d\n", X, Y);
  
  dim3 blockSize (BLOCKSIZE,BLOCKSIZE);
  dim3 gridSize (ceil( (float) X/BLOCKSIZE), ceil( (float) Y/BLOCKSIZE));

  SDL_Init(SDL_INIT_VIDEO);
  SDL_Surface *SDL_img = SDL_SetVideoMode(nb_pts_X, nb_pts_Y, 32, SDL_HWSURFACE | SDL_DOUBLEBUF);
  SDL_Event event;

  coord_t *h_bodies = (coord_t*) malloc(X*Y*sizeof(coord_t));

  coord_t *d_bodies_1, *d_bodies_2;
  hiprandState *states;
  
  hipMalloc(&d_bodies_1, X*Y*sizeof(coord_t));
  hipMalloc(&d_bodies_2, X*Y*sizeof(coord_t));
  hipMalloc(&states, X*Y*sizeof(hiprandState));

  init_coordinates_n_masses<<<gridSize, blockSize>>>(d_bodies_1, d_bodies_2, states, X,
                                                   max_X, max_Y, min_m, max_m, time(NULL));
  hipMemcpy(h_bodies, d_bodies_1, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
  update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
  SDL_Flip(SDL_img);
  for (int i=0; i<X*Y; i++) printf("%f %f %f | ", h_bodies[i].x, h_bodies[i].y, h_bodies[i].m); printf("\n");

  while (true)
  {
    update_bodies<<<gridSize, blockSize>>>(d_bodies_1, d_bodies_2, X, Y);
    hipMemcpy(h_bodies, d_bodies_2, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
    update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
    SDL_Flip(SDL_img);

    update_bodies<<<gridSize, blockSize>>>(d_bodies_2, d_bodies_1, X, Y);
    hipMemcpy(h_bodies, d_bodies_1, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
    //for (int i=0; i<X*Y; i++) printf("%f %f %f | ", h_bodies[i].x, h_bodies[i].y, h_bodies[i].m); printf("\n");
    update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
    SDL_Flip(SDL_img);
    if ( SDL_PollEvent(&event) )
      switch (event.type)
      {
        default :
          break;
        case SDL_KEYDOWN:
          switch (event.key.keysym.sym)
          {
            case SDLK_q :
              SDL_Quit();
              return EXIT_SUCCESS;
              break;
            default :
              break;
          }
      }
  }
}









