#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <SDL/SDL.h>

#define pot_const 1e-6
#define OFFSET 10
#define PI 3.141592653589793

typedef struct coord_t {float x, y, vx, vy, m;} coord_t;

__device__
coord_t potential(coord_t contrib, coord_t a, coord_t b)
{
  contrib.x += pot_const * b.m / powf( sqrtf( powf((a.x-b.x), 2) + powf(a.y-b.y, 2) ), 1)*(b.x-a.x);
  contrib.y += pot_const * b.m / powf( sqrtf( powf((a.x-b.x), 2) + powf(a.y-b.y, 2) ), 1)*(b.y-a.y);
  return contrib;
}

__device__
float calc_ray(coord_t * coord)
{
  return  sqrtf(powf(coord->x, 2) + powf(coord->y, 2));
}

__global__ 
void init_coordinates_n_masses(coord_t * bodies_1, coord_t * bodies_2, hiprandState_t * states, size_t X, 
                               float max_X, float max_Y, float init_ray, float min_m, float max_m, int seed)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  hiprand_init(seed, j*X+i, OFFSET, &states[j*X+i]);
  bodies_1[j*X+i].x = hiprand_normal(&states[j*X+i])*init_ray+max_X/2;
  bodies_1[j*X+i].y = hiprand_normal(&states[j*X+i])*init_ray+max_Y/2;
  //float ray = calc_ray(&bodies_1[j*X+i]); 
  bodies_1[j*X+i].vx = 50*(- bodies_1[j*X+i].y/max_Y + 1./2);
  bodies_1[j*X+i].vy = 50*(bodies_1[j*X+i].x/max_X - 1./2);
  bodies_1[j*X+i].m = (min_m+max_m)/2 + hiprand_normal(&states[j*X+i]) * (max_m-min_m)/2;
  bodies_2[j*X+i].m = bodies_1[j*X+i].m;
}

__global__ 
void update_bodies(coord_t * bodies_in, coord_t * bodies_out, size_t X, size_t Y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  
  coord_t contrib = {0,0};
  for(int h=0; h<Y; h++)
    for(int w=0; w<X; w++)
      if ( h != j && w != i)
        contrib = potential(contrib, bodies_in[j*X+i], bodies_in[h*X+w]);
  bodies_out[j*X+i].vx = bodies_in[j*X+i].vx + contrib.x;
  bodies_out[j*X+i].vy = bodies_in[j*X+i].vy + contrib.y;
  bodies_out[j*X+i].x = bodies_in[j*X+i].vx + bodies_in[j*X+i].x ;
  bodies_out[j*X+i].y = bodies_in[j*X+i].vy + bodies_in[j*X+i].y ;
}

void update_pixels(coord_t * h_bodies, Uint32 * pixels, float max_X, float max_Y, float min_m, float max_m, 
                   size_t X, size_t Y, size_t nb_pts_X, size_t nb_pts_Y)
{
  for (int t=0; t<nb_pts_X*nb_pts_Y; t++)
      pixels[t] = 0;

  size_t px, py;
  for (int y=0; y<Y; y++)
    for (int x=0; x<X; x++)
    {
      px = ceil( nb_pts_X * h_bodies[y*X+x].x / max_X);
      py = ceil( nb_pts_Y * h_bodies[y*X+x].y / max_Y);
      if (0<py && py<nb_pts_Y && 0<px && px< nb_pts_X)
        pixels[py*nb_pts_X+px] = floor( (16711680-65280) * (h_bodies[y*X+x].m - min_m) / max_m + 65280);
    }
}

int main(int argc, char * argv[])
{
  if (argc != 9)
  {
    printf("nb bodies, nb_pts_X, nb_pts_Y, max_X, max_Y, init_ray, min_m, max_m\n");
    return EXIT_SUCCESS;
  }

  size_t N = atoi(argv[1]),
         nb_pts_X = atoi(argv[2]),
         nb_pts_Y = atoi(argv[3]);
  float max_X = atof(argv[4]),
        max_Y = atof(argv[5]),
        init_ray = atof(argv[6]),
        min_m = atof(argv[7]),
        max_m = atof(argv[8]);

  size_t X, Y; 
  X = ceil(sqrt(N));
  Y = ceil((float) N/X);
  
  size_t blockside;
  if (N<=1024) blockside = 16;
  if ((N>1024) && (N<=4096)) blockside = 32;
  if (N>4096) blockside = 64;

  printf("X, Y, blockside = %d, %d, %d\n", X, Y, blockside);
    
  dim3 blockSize (blockside,blockside);
  dim3 gridSize (ceil( (float) X/blockside), ceil( (float) Y/blockside));

  SDL_Init(SDL_INIT_VIDEO);
  SDL_Surface *SDL_img = SDL_SetVideoMode(nb_pts_X, nb_pts_Y, 32, SDL_HWSURFACE | SDL_DOUBLEBUF);
  SDL_Event event;

  coord_t *h_bodies = (coord_t*) malloc(X*Y*sizeof(coord_t));

  coord_t *d_bodies_1, *d_bodies_2;
  hiprandState *states;
  
  hipMalloc(&d_bodies_1, X*Y*sizeof(coord_t));
  hipMalloc(&d_bodies_2, X*Y*sizeof(coord_t));
  hipMalloc(&states, X*Y*sizeof(hiprandState));

  init_coordinates_n_masses<<<gridSize, blockSize>>>(d_bodies_1, d_bodies_2, states, X,
                                                   max_X, max_Y, init_ray, min_m, max_m, time(NULL));
  hipMemcpy(h_bodies, d_bodies_1, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
  update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
  SDL_Flip(SDL_img);
  //for (int i=0; i<X*Y; i++) printf("%f %f %f | ", h_bodies[i].x, h_bodies[i].y, h_bodies[i].m); printf("\n");

  while (true)
  {
    update_bodies<<<gridSize, blockSize>>>(d_bodies_1, d_bodies_2, X, Y);
    hipMemcpy(h_bodies, d_bodies_2, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
    update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
    SDL_Flip(SDL_img);

    update_bodies<<<gridSize, blockSize>>>(d_bodies_2, d_bodies_1, X, Y);
    hipMemcpy(h_bodies, d_bodies_1, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
    //for (int i=0; i<X*Y; i++) printf("%f %f %f | ", h_bodies[i].x, h_bodies[i].y, h_bodies[i].m); printf("\n");
    update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
    SDL_Flip(SDL_img);
    if ( SDL_PollEvent(&event) )
      switch (event.type)
      {
        default :
          break;
        case SDL_KEYDOWN:
          switch (event.key.keysym.sym)
          {
            case SDLK_q :
              SDL_Quit();
              return EXIT_SUCCESS;
              break;
            case SDLK_r :
              init_coordinates_n_masses<<<gridSize, blockSize>>>(d_bodies_1, d_bodies_2, states, X,
                                                               max_X, max_Y, init_ray, min_m, max_m, time(NULL));
              hipMemcpy(h_bodies, d_bodies_1, X*Y*sizeof(coord_t), hipMemcpyDeviceToHost);
              update_pixels(h_bodies, (Uint32 *) SDL_img->pixels, max_X, max_Y, min_m, max_m, X, Y, nb_pts_X, nb_pts_Y);
              SDL_Flip(SDL_img);
              break;
            default :
              break;
          }
      }
  }
}









